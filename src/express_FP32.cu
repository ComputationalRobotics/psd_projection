#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>
#include <hipsolver.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <vector>
#include <iostream>

#include "psd_projection/express_FP32.h"
#include "psd_projection/lanczos.h"
#include "psd_projection/check.h"
#include "psd_projection/utils.h"

void express_FP32(
    hipblasHandle_t cublasH,
    double* mat,
    const int n,
    const int mat_offset
) {
    const int nn = n * n;

    /* Allocations */
    // device memory
    float *A, *A2, *A3;
    CHECK_CUDA( hipMalloc(&A,  nn * sizeof(float)) );
    CHECK_CUDA( hipMalloc(&A2, nn * sizeof(float)) );
    CHECK_CUDA( hipMalloc(&A3, nn * sizeof(float)) );

    // useful constants
    const float half       =  0.5f;
    const float minus_half = -0.5f;
    const float one        =  1.0f;
    const float one_n_half =  1.5f;
    const float zero       =  0.0f;

    /* Convert the initial matrix*/
    convert_double_to_float(mat + mat_offset, A, nn);

    /* Coefficients */
    // std::vector<std::vector<float>> coeff = {
    //     {8.4724206924, -24.5001735687, 17.7268180847},
    //     {4.2052841187, -3.0549299717, 0.5567536354},
    //     {4.0443077087, -2.9473149776, 0.5449726582},
    //     {3.5078327656, -2.5842490196, 0.5067413449},
    //     {2.5075511932, -1.8485442400, 0.4358045161}
    // };
    std::vector<std::vector<float>> coeff = { 
        { 8.3885353390, -23.7796270883, 16.8664591580 }, 
        { 4.1636476423, -2.9650849331, 0.5297319805 }, 
        { 4.0042650581, -2.8606348801, 0.5185227850 }, 
        { 3.4731017481, -2.5082466382, 0.4821470022 }, 
        { 2.4827239537, -1.7941788274, 0.4146530436 }, 
    };

    /* Approximation of the step function */
    for (int i = 0; i < coeff.size(); i++) {
        const float a = coeff[i][0];
        const float b = coeff[i][1];
        const float c = coeff[i][2];

        /* Compute the powers of A*/
        // A2 = A * A
        CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A, n, A, n, &zero, A2, n) );

        // A3 = A2 * A
        CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A2, n, A, n, &zero, A3, n) );

        // A = a * A
        CHECK_CUBLAS( hipblasSscal(cublasH, nn, &a, A, 1) );
        // A = b * A3 + A
        CHECK_CUBLAS( hipblasSaxpy(cublasH, nn, &b, A3, 1, A, 1) );
        // at this point, A = a * A + b * A3

        // A = c * A3 * A2 + A
        CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &c, A3, n, A2, n, &one, A, n) );

        /* Symmetrize A */
        symmetrizeFloat(cublasH, A, n, A2); // we use A2 as a workspace
    }

    /* Smoothing function */
    for (int i = 0; i < 3; i++) {
        // A2 = A * A
        CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A, n, A, n, &zero, A2, n) );

        // A3 = A2 * A
        CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A2, n, A, n, &zero, A3, n) );

        /* Symmetrize A3 */
        symmetrizeFloat(cublasH, A3, n, A2); // we use A2 as a workspace

        /* Compute A = 1.5 * A - 0.5 * A3 */
        // A = 1.5 * A
        CHECK_CUBLAS( hipblasSscal(cublasH, nn, &one_n_half, A, 1) );
        // A = -0.5 * A3 + A
        CHECK_CUBLAS( hipblasSaxpy(cublasH, nn, &minus_half, A3, 1, A, 1) );

        /* Symmetrize A */
        symmetrizeFloat(cublasH, A, n, A2); // we use A2 as a workspace
    }

    /* Compute A = (I + A)/2 */
    // build I on device and store it in A2
    build_identity(cublasH, A2, n, 1024);

    // A = 1 * I + A
    CHECK_CUBLAS( hipblasSaxpy(cublasH, nn, &one, A2, 1, A, 1) );
    // A = 0.5 * A
    CHECK_CUBLAS( hipblasSscal(cublasH, nn, &half, A, 1) );

    /* Symmetrize A */
    symmetrizeFloat(cublasH, A, n, A2); // we use A2 as a workspace

    /* Multiply the original matrix by A */
    // W = A_origin * A
    convert_double_to_float(mat + mat_offset, A2, nn);
    CHECK_CUBLAS( hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A2, n, A, n, &zero, A3, n) );

    /* Symmetrize W */
    symmetrizeFloat(cublasH, A3, n, A2); // we use A2 as a workspace

    /* Copy the result back to mat */
    convert_float_to_double(A3, mat + mat_offset, nn);

    /* Free device memory */
    CHECK_CUDA( hipFree(A) );
    CHECK_CUDA( hipFree(A2) );
    CHECK_CUDA( hipFree(A3) );
}

void express_FP32_auto_scale(
    hipblasHandle_t cublasH,
    hipsolverHandle_t solverH,
    double* mat,
    const int n,
    const int mat_offset
) {
    size_t nn = n * n;
    
    // Use the Lanczos method to approximate the two-norm of the matrix
    double lo, up;
    approximate_two_norm(
        cublasH, solverH, mat + mat_offset, n, &lo, &up
    );

    // scale to have eigenvalues in [-1, 1]
    const double scale = up > 0.0 ? up : 1.0;
    const double inv_scale = 1.0/scale;
    CHECK_CUBLAS( hipblasDscal(cublasH, nn, &inv_scale, mat + mat_offset, 1) );

    // project the matrix using the express_FP32 function
    express_FP32(
        cublasH, mat + mat_offset, n, 0
    );

    // rescale the result back to the original scale
    CHECK_CUBLAS( hipblasDscal(cublasH, nn, &scale,  mat + mat_offset, 1) );
}

void express_FP32_auto_scale_deflate(
    hipblasHandle_t cublasH,
    hipsolverHandle_t solverH,
    double* mat,
    const int n,
    const int mat_offset,
    const size_t k,
    const double tol,
    const double ortho_tol
) {
    size_t nn = n * n;
    
    /* Step 1: compute the largest eigenpairs of the matrix */
    size_t r;
    double *eigenvalues, *eigenvectors;
    CHECK_CUDA( hipMalloc(&eigenvalues,      k * sizeof(double)) );
    CHECK_CUDA( hipMalloc(&eigenvectors, n * k * sizeof(double)) );

    double _ = compute_eigenpairs(
        cublasH, solverH, mat + mat_offset, n, k, &r, eigenvalues, eigenvectors, 0, tol, ortho_tol
    );

    std::vector<double> eigenvalues_host(r);
    CHECK_CUDA( hipMemcpy(eigenvalues_host.data(), eigenvalues, r * sizeof(double), D2H) );

    /* Step 2: remove the largest eigenvalues from the matrix */
    for (int i = 0; i < r; i++) {
        // X <- X - \lambda_i * v_i v_i^T
        double lambda = -eigenvalues_host[i];
        double *v_i = eigenvectors + i * n;
        CHECK_CUBLAS( hipblasDger(cublasH, n, n, &lambda, v_i, 1, v_i, 1, mat + mat_offset, n) );
    }

    /* Step 3: scale the deflated matrix */
    // size_t r2;
    // double up = compute_eigenpairs(
    //     cublasH, solverH, mat + mat_offset, n, 0, &r2, eigenvalues, eigenvectors, 0, tol, ortho_tol
    // );
    double up, lo;
    approximate_two_norm(
        cublasH, solverH, mat + mat_offset, n, &lo, &up
    );

    // scale to have eigenvalues in [-1, 1]
    const double scale = up > 0.0 ? up : 1.0;
    const double inv_scale = 1.0/scale;
    CHECK_CUBLAS( hipblasDscal(cublasH, nn, &inv_scale, mat + mat_offset, 1) );

    /* Step 4: project the matrix using the express_FP32 function */
    express_FP32(
        cublasH, mat + mat_offset, n, 0
    );

    /* Step 5: rescale the matrix back and add the deflated eigenvalues back */
    CHECK_CUBLAS( hipblasDscal(cublasH, nn, &scale, mat + mat_offset, 1) );

    for (int i = 0; i < r; i++) {
        // X <- X + \lambda_i * v_i v_i^T
        double lambda = eigenvalues_host[i];
        if (lambda > 0.0) { // only add positive eigenvalues
            double *v_i = eigenvectors + i * n;
            CHECK_CUBLAS( hipblasDger(cublasH, n, n, &lambda, v_i, 1, v_i, 1, mat + mat_offset, n) );
        }
    }
}