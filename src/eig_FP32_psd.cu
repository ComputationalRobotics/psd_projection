#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iomanip>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <random>

#include "psd_projection/check.h"
#include "psd_projection/utils.h"
#include "psd_projection/eig_FP32_psd.h"

double* eig_FP32_psd(hipsolverHandle_t solverH, hipblasHandle_t cublasH, double* dA, size_t n, bool return_eigenvalues) {
    size_t nn = n * n;
    float one_s = 1.0;
    float zero_s = 0.0;
    
    int *devInfo; CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    float *sA;
    CHECK_CUDA(hipMalloc(&sA, nn*sizeof(float)));
    
    // convert dA from double to float
    convert_double_to_float(dA, sA, nn);

    float *sW; CHECK_CUDA(hipMalloc(&sW, n*sizeof(float)));
    int lwork_ev = 0;
    CHECK_CUSOLVER(hipsolverDnSsyevd_bufferSize(
        solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
        n, sA, n, sW, &lwork_ev));
    float *sWork_ev; CHECK_CUDA(hipMalloc(&sWork_ev, lwork_ev*sizeof(float)));
    CHECK_CUSOLVER(hipsolverDnSsyevd(
        solverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_UPPER,
        n, sA, n, sW,
        sWork_ev, lwork_ev, devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    std::vector<float> W_h(n);
    CHECK_CUDA(hipMemcpy(W_h.data(), sW, n*sizeof(float), D2H));
    for (int i = 0; i < n; i++) {
        if (W_h[i] < 0)
            W_h[i] = 0;
    }

    // Copy eigenvectors from dA to dV
    float *sV; CHECK_CUDA(hipMalloc(&sV, nn*sizeof(float)));
    CHECK_CUDA(hipMemcpy(sV, sA, nn*sizeof(float), D2D));

    // Scale columns of dV by W_h
    for (int i = 0; i < n; i++)
        CHECK_CUBLAS(hipblasSscal(cublasH, n, &W_h[i], sV + i*n, 1));

    // Reconstruct A_psd = V * V^T
    float *sTmp; CHECK_CUDA(hipMalloc(&sTmp, nn*sizeof(float)));
    CHECK_CUBLAS(hipblasGemmEx(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, n, n,
        &one_s,
        sV, HIP_R_32F, n,
        sA, HIP_R_32F, n,
        &zero_s,
        sTmp, HIP_R_32F, n,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipMemcpy(sA, sTmp, nn*sizeof(float), D2D));

    convert_float_to_double(sA, dA, nn);

    // Cleanup
    CHECK_CUDA(hipFree(sWork_ev));
    CHECK_CUDA(hipFree(sA));
    CHECK_CUDA(hipFree(sTmp));
    CHECK_CUDA(hipFree(sV));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    if (!return_eigenvalues) {
        CHECK_CUDA(hipFree(sW));
        return nullptr;
    } else {
        double *dW; CHECK_CUDA(hipMalloc(&dW, n*sizeof(double)));
        convert_float_to_double(sW, dW, n);
        CHECK_CUDA(hipFree(sW));
        return dW;
    }
}