#include "hip/hip_runtime.h"
#include "psd_projection/utils.h"
#include "psd_projection/check.h"
#include <hipblas.h>
#include <random>
#include <chrono>

void symmetrizeFloat(
    hipblasHandle_t cublasH, float* M, int n, float* workspace
) {
    const float one = 1.0, half = 0.5, zero = 0.0;

    // workspace = M^T
    CHECK_CUBLAS(hipblasSgeam(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
        n, n,
        &one, M, n,
        &zero, M, n,
        workspace, n
    ));

    // M = M + workspace (which is M^T)
    CHECK_CUBLAS(hipblasSgeam(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n,
        &one, M, n,
        &one, workspace, n,
        M, n
    ));

    // M = 0.5 * M
    CHECK_CUBLAS(hipblasSscal(cublasH, n * n, &half, M, 1));
}

__global__ void convert_double_to_float_kernel(const double* in, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<float>(in[idx]);
    }
}

__global__ void convert_float_to_double_kernel(const float* in, double* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<double>(in[idx]);
    }
}

void convert_double_to_float(const double* d_in, float* d_out, int n, const int threadsPerBlock) {
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    convert_double_to_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);
    hipDeviceSynchronize();
}

void convert_float_to_double(const float* d_in, double* d_out, int n, const int threadsPerBlock) {
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    convert_float_to_double_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);
    hipDeviceSynchronize();
}

__global__ void build_identity_kernel(float* mat, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n)
        mat[idx] = (idx / n == idx % n) ? 1.0f : 0.0f;
}

void build_identity(
    hipblasHandle_t cublasH,
    float* mat,
    int n,
    const int threadsPerBlock
) {
    const int blocksPerGrid = (n * n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to build identity matrix
    build_identity_kernel<<<blocksPerGrid, threadsPerBlock>>>(mat, n);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void add_identity_kernel(float* mat, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col) {
            mat[idx] += 1.0f; // Add 1 to the diagonal elements
        }
    }
}

void add_identity(
    hipblasHandle_t cublasH,
    float* mat,
    int n
) {
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = (n * n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to add identity matrix
    add_identity_kernel<<<blocksPerGrid, threadsPerBlock>>>(mat, n);
    CHECK_CUDA(hipGetLastError());
}

__global__ void float4_to_half_kernel(
    const float4* __restrict__ A4,
    __half2 * __restrict__ B2,
    size_t N4
) {
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= N4) return;

    // load 4 floats
    float4 v = A4[idx];

    // pack low two floats into half2
    B2[2*idx + 0] = __float22half2_rn(make_float2(v.x, v.y));
    // pack high two floats into half2
    B2[2*idx + 1] = __float22half2_rn(make_float2(v.z, v.w));
}

void convert_float_to_half4(const float* dA, __half* dB, size_t N) {
    size_t N4 = (N + 3)/4;  // how many float4’s
    auto A4 = reinterpret_cast<const float4*>(dA);
    auto B2 = reinterpret_cast<__half2*>(dB);

    const int blk = 1024;
    int grid = (N4 + blk - 1)/blk;
    float4_to_half_kernel<<<grid,blk>>>(A4, B2, N4);
}

// Kernel to replace A by I - A
__global__ void identity_minus_kernel(const float* A_in, float* A_out, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col) {
            A_out[idx] = 1.0f - A_in[idx]; // diagonal elements
        } else {
            A_out[idx] = -A_in[idx]; // off-diagonal elements
        }
    }
}

void identity_minus(
    const float* A_in,
    float* A_out,
    const int n
) {
    const int nn = n * n;
    const int threads = 1024;
    const int blocks = (nn + threads - 1) / threads;

    // Launch kernel to compute I - A
    identity_minus_kernel<<<blocks, threads>>>(A_in, A_out, n);
    CHECK_CUDA(hipGetLastError());
}

// Kernel to replace A by I + A
__global__ void identity_plus_kernel(const float* A_in, float* A_out, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col)
            A_out[idx] = 1.0f + A_in[idx]; // diagonal elements
        else
            A_out[idx] = A_in[idx]; // off-diagonal elements
    }
}

void identity_plus(
    const float* A_in, // device pointer to matrix A
    float* A_out, // device pointer to matrix A
    const int n
) {
    const int nn = n * n;
    const int threads = 1024;
    const int blocks = (nn + threads - 1) / threads;

    // Launch kernel to compute I + A
    identity_plus_kernel<<<blocks, threads>>>(A_in, A_out, n);
    CHECK_CUDA(hipGetLastError());
}

unsigned long make_seed() {
    std::random_device rd;

    std::seed_seq seq{
        rd(), rd(), rd(), rd(),
        static_cast<unsigned>(std::chrono::high_resolution_clock::now()
                              .time_since_epoch().count())   // mixes in time
    };

    std::mt19937_64 mixer(seq);   // 64-bit Mersenne Twister
    return mixer();               // one well-mixed 64-bit value
}

__global__ void fill_random_kernel(double* vec, int n, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        vec[idx] = hiprand_uniform_double(&state); // random double in (0,1]
    }
}

void fill_random(double* vec, int n, unsigned long seed, const int threadsPerBlock) {
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    fill_random_kernel<<<blocks, threadsPerBlock>>>(vec, n, seed);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in fill_random: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}