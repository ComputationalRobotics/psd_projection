#include "hip/hip_runtime.h"
#include "psd_projection/utils.h"
#include "psd_projection/check.h"
#include <hipblas.h>

void symmetrizeFloat(
    hipblasHandle_t cublasH, float* M, int n, float* workspace
) {
    const float one = 1.0, half = 0.5, zero = 0.0;

    // workspace = M^T
    CHECK_CUBLAS(hipblasSgeam(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
        n, n,
        &one, M, n,
        &zero, M, n,
        workspace, n
    ));

    // M = M + workspace (which is M^T)
    CHECK_CUBLAS(hipblasSgeam(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n,
        &one, M, n,
        &one, workspace, n,
        M, n
    ));

    // M = 0.5 * M
    CHECK_CUBLAS(hipblasSscal(cublasH, n * n, &half, M, 1));
}

__global__ void convert_double_to_float_kernel(const double* in, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<float>(in[idx]);
    }
}

__global__ void convert_float_to_double_kernel(const float* in, double* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<double>(in[idx]);
    }
}

void convert_double_to_float(const double* d_in, float* d_out, int n, const int threadsPerBlock) {
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    convert_double_to_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);
    hipDeviceSynchronize();
}

void convert_float_to_double(const float* d_in, double* d_out, int n, const int threadsPerBlock) {
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    convert_float_to_double_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);
    hipDeviceSynchronize();
}

__global__ void build_identity_kernel(float* mat, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n)
        mat[idx] = (idx / n == idx % n) ? 1.0f : 0.0f;
}

void build_identity(
    hipblasHandle_t cublasH,
    float* mat,
    int n,
    const int threadsPerBlock
) {
    const int blocksPerGrid = (n * n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to build identity matrix
    build_identity_kernel<<<blocksPerGrid, threadsPerBlock>>>(mat, n);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void add_identity_kernel(float* mat, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col) {
            mat[idx] += 1.0f; // Add 1 to the diagonal elements
        }
    }
}

void add_identity(
    hipblasHandle_t cublasH,
    float* mat,
    int n
) {
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = (n * n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel to add identity matrix
    add_identity_kernel<<<blocksPerGrid, threadsPerBlock>>>(mat, n);
    CHECK_CUDA(hipGetLastError());
}

__global__ void float4_to_half_kernel(
    const float4* __restrict__ A4,
    __half2 * __restrict__ B2,
    size_t N4
) {
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= N4) return;

    // load 4 floats
    float4 v = A4[idx];

    // pack low two floats into half2
    B2[2*idx + 0] = __float22half2_rn(make_float2(v.x, v.y));
    // pack high two floats into half2
    B2[2*idx + 1] = __float22half2_rn(make_float2(v.z, v.w));
}

void convert_float_to_half4(const float* dA, __half* dB, size_t N) {
    size_t N4 = (N + 3)/4;  // how many float4’s
    auto A4 = reinterpret_cast<const float4*>(dA);
    auto B2 = reinterpret_cast<__half2*>(dB);

    const int blk = 1024;
    int grid = (N4 + blk - 1)/blk;
    float4_to_half_kernel<<<grid,blk>>>(A4, B2, N4);
}

// Kernel to replace A by I - A
__global__ void identity_minus_kernel(const float* A_in, float* A_out, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col) {
            A_out[idx] = 1.0f - A_in[idx]; // diagonal elements
        } else {
            A_out[idx] = -A_in[idx]; // off-diagonal elements
        }
    }
}

void identity_minus(
    const float* A_in,
    float* A_out,
    const int n
) {
    const int nn = n * n;
    const int threads = 1024;
    const int blocks = (nn + threads - 1) / threads;

    // Launch kernel to compute I - A
    identity_minus_kernel<<<blocks, threads>>>(A_in, A_out, n);
    CHECK_CUDA(hipGetLastError());
}

// Kernel to replace A by I + A
__global__ void identity_plus_kernel(const float* A_in, float* A_out, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        if (row == col)
            A_out[idx] = 1.0f + A_in[idx]; // diagonal elements
        else
            A_out[idx] = A_in[idx]; // off-diagonal elements
    }
}

void identity_plus(
    const float* A_in, // device pointer to matrix A
    float* A_out, // device pointer to matrix A
    const int n
) {
    const int nn = n * n;
    const int threads = 1024;
    const int blocks = (nn + threads - 1) / threads;

    // Launch kernel to compute I + A
    identity_plus_kernel<<<blocks, threads>>>(A_in, A_out, n);
    CHECK_CUDA(hipGetLastError());
}