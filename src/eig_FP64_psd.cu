#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iomanip>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <random>

#include "psd_projection/check.h"
#include "psd_projection/utils.h"
#include "psd_projection/eig_FP64_psd.h"
#include "psd_projection/lopbcg.h"

double* eig_FP64_psd(hipsolverHandle_t solverH, hipblasHandle_t cublasH, double* dA, size_t n, bool return_eigenvalues) {
    int *devInfo; CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    size_t nn = n * n;
    double one_d = 1.0;
    double zero_d = 0.0;

    double *dW, *dW_out;
    CHECK_CUDA(hipMalloc(&dW, n*sizeof(double)));

    int lwork_ev = 0;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(
        solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW, &lwork_ev));
    double *dWork_ev; CHECK_CUDA(hipMalloc(&dWork_ev, lwork_ev*sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDsyevd(
        solverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW,
        dWork_ev, lwork_ev, devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    if (return_eigenvalues) { // save the eigevalues before zeroing them out
        CHECK_CUDA(hipMalloc(&dW_out, n*sizeof(double)));
        CHECK_CUDA(hipMemcpy(dW_out, dW, n*sizeof(double), hipMemcpyDeviceToDevice));
    }

    max_dense_vector_zero(dW, n);

    // Copy eigenvectors from dA to dV
    double *dV; CHECK_CUDA(hipMalloc(&dV, nn*sizeof(double)));
    CHECK_CUDA(hipMemcpy(dV, dA, nn*sizeof(double), hipMemcpyDeviceToDevice));

    
    // Scale columns of dV by W_h
    CHECK_CUBLAS(hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE));
    for(int i = 0; i < n; i++){
        CHECK_CUBLAS(hipblasDscal(cublasH, n, &dW[i], dV + i*n, 1));
    }
    CHECK_CUBLAS(hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST));

    // Reconstruct A_psd = V * V^T
    double *dTmp; CHECK_CUDA(hipMalloc(&dTmp, nn*sizeof(double)));
    CHECK_CUBLAS(hipblasGemmEx(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, n, n,
        &one_d,
        dV, HIP_R_64F, n,
        dA, HIP_R_64F, n,
        &zero_d,
        dTmp, HIP_R_64F, n,
        HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipMemcpy(dA, dTmp, nn*sizeof(double), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(dTmp));
    CHECK_CUDA(hipFree(dW));
    CHECK_CUDA(hipFree(dV));
    CHECK_CUDA(hipFree(dWork_ev));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipDeviceSynchronize());
    
    return dW_out;
}

void eig_FP64_deflate(
    hipsolverHandle_t solverH, 
    hipblasHandle_t cublasH,
    double* mat,
    size_t n,
    const size_t k,
    const int maxiter,
    const double tol,
    const bool verbose
) {
    size_t nn = n * n;
    double minus_one = -1.0;

    // TODO: use a workspace for the eigenvalues and eigenvectors
    double *eigenvalues_max, *eigenvectors_max;
    double *eigenvalues_min, *eigenvectors_min;
    CHECK_CUDA( hipMalloc(&eigenvalues_max,      k * sizeof(double)) );
    CHECK_CUDA( hipMalloc(&eigenvectors_max, n * k * sizeof(double)) );
    CHECK_CUDA( hipMalloc(&eigenvalues_min,      k * sizeof(double)) );
    CHECK_CUDA( hipMalloc(&eigenvectors_min, n * k * sizeof(double)) );

    /* Step 1: compute the largest eigenpairs of the matrix */
    lopbcg(
        cublasH, solverH, mat, eigenvectors_max, eigenvalues_max, n, k, false, maxiter, tol, verbose
    );
    // negate the eigenvalues
    CHECK_CUBLAS(hipblasDscal(cublasH, k, &minus_one, eigenvalues_max, 1));

    /* Step 2: remove the largest eigenvalues from the matrix */
    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE);
    for (int i = 0; i < k; i++) {
        // X <- X - \lambda_i * v_i v_i^T
        double *v_i = eigenvectors_max + i * n;
        double *m_lambda_i = eigenvalues_max + i;
        CHECK_CUBLAS( hipblasDger(cublasH, n, n, m_lambda_i, v_i, 1, v_i, 1, mat, n) );
    }
    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);

    /* Step 1bis: compute the lowest eigenpairs of the matrix */
    // change the matrix sign to reuse LOPBCG code
    CHECK_CUBLAS(hipblasDscal(cublasH, nn, &minus_one, mat, 1));
    lopbcg(
        cublasH, solverH, mat, eigenvectors_min, eigenvalues_min, n, k, false, maxiter, tol, verbose
    );
    // note: the min eigenvalues are already negated since we used -A

    /* Step 2bis: remove the lowest eigenvalues from the matrix */
    // restore the matrix sign
    CHECK_CUBLAS(hipblasDscal(cublasH, nn, &minus_one, mat, 1));
    // remove them
    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE);
    for (int i = 0; i < k; i++) {
        // X <- X - \lambda_i * v_i v_i^T
        double *v_i = eigenvectors_min + i * n;
        double *m_lambda_i = eigenvalues_min + i;
        CHECK_CUBLAS( hipblasDger(cublasH, n, n, m_lambda_i, v_i, 1, v_i, 1, mat, n) );
    }
    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);

    /* Step 3: project the matrix using the eig_FP64_psd function */
    eig_FP64_psd(solverH, cublasH, mat, n);

    /* Step 4: add back the eigenvalues */
    // add only positive eigenvalues back
    CHECK_CUBLAS( hipblasDscal(cublasH, k, &minus_one, eigenvalues_max, 1) );
    CHECK_CUBLAS( hipblasDscal(cublasH, k, &minus_one, eigenvalues_min, 1) );
    max_dense_vector_zero(eigenvalues_max, k);
    max_dense_vector_zero(eigenvalues_min, k);

    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE);
    for (int i = 0; i < k; i++) {
        // X <- X + \lambda_i * v_i v_i^T
        double *m_lambda_i = eigenvalues_max + i;
        double *v_i = eigenvectors_max + i * n;
        CHECK_CUBLAS( hipblasDger(cublasH, n, n, m_lambda_i, v_i, 1, v_i, 1, mat, n) );
    }
    for (int i = 0; i < k; i++) {
        // X <- X + \lambda_i * v_i v_i^T
        double *m_lambda_i = eigenvalues_min + i;
        double *v_i = eigenvectors_min + i * n;
        CHECK_CUBLAS( hipblasDger(cublasH, n, n, m_lambda_i, v_i, 1, v_i, 1, mat, n) );
    }
    hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST);

    /* Free device memory */
    CHECK_CUDA( hipFree(eigenvalues_max) );
    CHECK_CUDA( hipFree(eigenvectors_max) );
    CHECK_CUDA( hipFree(eigenvalues_min) );
    CHECK_CUDA( hipFree(eigenvectors_min) );
}
