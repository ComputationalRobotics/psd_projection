#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iomanip>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <random>

#include "psd_projection/check.h"
#include "psd_projection/utils.h"
#include "psd_projection/eig_FP64_psd.h"

double* eig_FP64_psd(hipsolverHandle_t solverH, hipblasHandle_t cublasH, double* dA, size_t n, bool return_eigenvalues) {
    int *devInfo; CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    size_t nn = n * n;
    double one_d = 1.0;
    double zero_d = 0.0;

    double *dW, *dW_out;
    CHECK_CUDA(hipMalloc(&dW, n*sizeof(double)));

    int lwork_ev = 0;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(
        solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW, &lwork_ev));
    double *dWork_ev; CHECK_CUDA(hipMalloc(&dWork_ev, lwork_ev*sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDsyevd(
        solverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW,
        dWork_ev, lwork_ev, devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    if (return_eigenvalues) { // save the eigevalues before zeroing them out
        CHECK_CUDA(hipMalloc(&dW_out, n*sizeof(double)));
        CHECK_CUDA(hipMemcpy(dW_out, dW, n*sizeof(double), hipMemcpyDeviceToDevice));
    }

    max_dense_vector_zero(dW, n);

    // Copy eigenvectors from dA to dV
    double *dV; CHECK_CUDA(hipMalloc(&dV, nn*sizeof(double)));
    CHECK_CUDA(hipMemcpy(dV, dA, nn*sizeof(double), hipMemcpyDeviceToDevice));

    
    // Scale columns of dV by W_h
    CHECK_CUBLAS(hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE));
    for(int i = 0; i < n; i++){
        CHECK_CUBLAS(hipblasDscal(cublasH, n, &dW[i], dV + i*n, 1));
    }
    CHECK_CUBLAS(hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_HOST));

    // Reconstruct A_psd = V * V^T
    double *dTmp; CHECK_CUDA(hipMalloc(&dTmp, nn*sizeof(double)));
    CHECK_CUBLAS(hipblasGemmEx(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, n, n,
        &one_d,
        dV, HIP_R_64F, n,
        dA, HIP_R_64F, n,
        &zero_d,
        dTmp, HIP_R_64F, n,
        HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipMemcpy(dA, dTmp, nn*sizeof(double), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(dTmp));
    CHECK_CUDA(hipFree(dW));
    CHECK_CUDA(hipFree(dV));
    CHECK_CUDA(hipFree(dWork_ev));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipDeviceSynchronize());
    
    return dW_out;
}