#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iomanip>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <random>

#include "psd_projection/check.h"
#include "psd_projection/utils.h"
#include "psd_projection/eig_FP64_psd.h"

void eig_FP64_psd(hipsolverHandle_t solverH, hipblasHandle_t cublasH, double* dA, size_t n) {
    int *devInfo; CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    size_t nn = n * n;
    double one_d = 1.0;
    double zero_d = 0.0;

    double *dW; CHECK_CUDA(hipMalloc(&dW, n*sizeof(double)));
    int lwork_ev = 0;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(
        solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW, &lwork_ev));
    double *dWork_ev; CHECK_CUDA(hipMalloc(&dWork_ev, lwork_ev*sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDsyevd(
        solverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW,
        dWork_ev, lwork_ev, devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    std::vector<double> W_h(n);
    CHECK_CUDA(hipMemcpy(W_h.data(), dW, n*sizeof(double), hipMemcpyDeviceToHost));

    // std::cout << "Max and min eigenvalues: "
    //           << *std::max_element(W_h.begin(), W_h.end()) << ", "
    //           << *std::min_element(W_h.begin(), W_h.end()) << std::endl;

    for(int i=0;i<n;i++) if(W_h[i]<0) W_h[i]=0;

    // Copy eigenvectors from dA to dV
    double *dV; CHECK_CUDA(hipMalloc(&dV, nn*sizeof(double)));
    CHECK_CUDA(hipMemcpy(dV, dA, nn*sizeof(double), hipMemcpyDeviceToDevice));

    // Scale columns of dV by W_h
    for(int i=0;i<n;i++){
        CHECK_CUBLAS(hipblasDscal(cublasH, n, &W_h[i], dV + i*n, 1));
    }

    // Reconstruct A_psd = V * V^T
    double *dTmp; CHECK_CUDA(hipMalloc(&dTmp, nn*sizeof(double)));
    CHECK_CUBLAS(hipblasGemmEx(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, n, n,
        &one_d,
        dV, HIP_R_64F, n,
        dA, HIP_R_64F, n,
        &zero_d,
        dTmp, HIP_R_64F, n,
        HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipMemcpy(dA, dTmp, nn*sizeof(double), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(dTmp));
    CHECK_CUDA(hipFree(dV));
    CHECK_CUDA(hipFree(dWork_ev));
    CHECK_CUDA(hipFree(dW));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    return;
}