#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <vector>
#include <cassert>
#include <cstdio>

#include "psd_projection/lopbcg.h"
#include "psd_projection/check.h"
#include "psd_projection/utils.h"

__global__ void reverse_vector_kernel(const double* in, double* out, int m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m) {
        out[idx] = in[m - 1 - idx];
    }
}

void reverse_vector(const double* in, double* out, int m) {
    int threads = 1024;
    int blocks = (m + threads - 1) / threads;
    reverse_vector_kernel<<<blocks, threads>>>(in, out, m);
    CHECK_CUDA(hipGetLastError());
}

__global__ void reverse_columns_kernel(const double* in, double* out, int n, int m) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < n && col < m) {
        // Copy column col to column (m - 1 - col)
        out[row + (m - 1 - col) * n] = in[row + col * n];
    }
}

void reverse_columns(const double* in, double* out, int n, int m) {
    dim3 threads(32, 32);
    dim3 blocks((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
    reverse_columns_kernel<<<blocks, threads>>>(in, out, n, m);
    CHECK_CUDA(hipGetLastError());
}

void lopbcg(
    const double* A, // n x n, device pointer
    double* V,       // n x m, device pointer (output eigenvectors)
    double* D,       // m x m, device pointer (output eigenvalues, diagonal)
    const int n,
    const int m ,      // number of eigenpairs
    const int maxiter, // maximum iterations
    const double tol,   // convergence tolerance
    const bool verbose
) {
    assert(m > 0);
    assert(n > 0);
    assert(3*m <= n);

    /* Allocations */
    // cuBLAS/cuSOLVER handles
    hipblasHandle_t cublasH = nullptr;
    hipsolverHandle_t cusolverH = nullptr;
    CHECK_CUBLAS(hipblasCreate(&cublasH));
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    // allocate the device memory
    double *X_k, *X_k_tmp, *Lam_k, *Lam_k_tmp, *T, *Tt, *Delta_X_k, *T_tmp, *R_k;
    double *XRD, *Lam_all, *XRD_tmp, *T_XRD, *Tt_XRD, *T_tmp_XRD;
    CHECK_CUDA(hipMalloc(&X_k,            n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&X_k_tmp,        n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_k,              m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_k_tmp,          m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Delta_X_k,      n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_tmp,          n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T,              m * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Tt,             m * m * sizeof(double)));
    
    CHECK_CUDA(hipMalloc(&XRD,          n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_all,          3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&XRD_tmp,      n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_XRD,      3*m * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Tt_XRD,     3*m * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_tmp_XRD,    n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&R_k,          n * 3*m * sizeof(double)));

    double norm_R_k;

    // workspace for QR decomposition of X_k
    int lwork, *devInfo;
    double *d_work, *tau;
    CHECK_CUDA(hipMalloc(&tau, m * sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, n, m, X_k, n, &lwork));
    CHECK_CUDA(hipMalloc(&d_work, lwork * sizeof(double)));
    CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));

    // workspace for QR decomposition of XRD
    int lwork_xrd;
    double *d_work_xrd, *tau_xrd;
    CHECK_CUDA(hipMalloc(&tau_xrd, 3*m * sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, n, 3*m, XRD, n, &lwork_xrd));
    CHECK_CUDA(hipMalloc(&d_work_xrd, lwork_xrd * sizeof(double)));

    // workspace for eigenvalue decomposition
    int lwork_eig;
    double *d_work_eig;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                                m, T, m, Lam_k, &lwork_eig));
    CHECK_CUDA(hipMalloc(&d_work_eig, lwork_eig * sizeof(double)));

    // workspace for eigenvalue decomposition of XRD
    int lwork_eig_XRD;
    double *d_work_eig_XRD;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                                3*m, T_XRD, 3*m, Lam_all, &lwork_eig_XRD));
    CHECK_CUDA(hipMalloc(&d_work_eig_XRD, lwork_eig_XRD * sizeof(double)));

    // useful constants
    const double one = 1.0;
    const double zero = 0.0;
    const double half = 0.5;
    const double neg1 = -1.0;

    /* Initialization of X_k */
    fill_random(X_k, n * m, make_seed());

    // compute QR factorization (X_k overwritten with R, tau contains Householder scalars)
    CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, n, m, X_k, n, tau, d_work, lwork, devInfo));

    // generate Q from the result (X_k overwritten with Q)
    CHECK_CUSOLVER(hipsolverDnDorgqr(cusolverH, n, m, m, X_k, n, tau, d_work, lwork, devInfo));

    /* Compute new X_k using T */
    // T = Q^T * A * Q
    // T_tmp = Q^T * A
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, n,
                             &one, X_k, n, A, n,
                             &zero, T_tmp, m));
    // T = T_tmp * Q
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, n,
                             &one, T_tmp, m, X_k, n,
                             &zero, T, m));
    // copy T to Tt
    CHECK_CUBLAS(hipblasDcopy(cublasH, m * m, T, 1, Tt, 1));

    // T = 0.5 * (T + T^T)
    CHECK_CUBLAS(hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m,
                             &half, T, m,
                             &half, Tt, m,
                             T, m));

    // compute eigenvalues and eigenvectors of T
    // both are in increasing order
    CHECK_CUSOLVER(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                    m, T, m, Lam_k_tmp, d_work_eig, lwork_eig, devInfo));
    // reverse Lam_k_tmp to get Lam_k in decreasing order
    reverse_vector(Lam_k_tmp, Lam_k, m);

    // X_k = Q * T
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                             &one, X_k, n, T, m,
                             &zero, X_k, n));

    // Delta_X_k = X_k
    CHECK_CUBLAS(hipblasDcopy(cublasH, n * m, X_k, 1, Delta_X_k, 1));

    for (int iter = 1; iter <= maxiter; iter++) {
        // R_k = A * X_k - X_k * Lam_k
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n,
                                 &one, A, n, X_k, n,
                                 &zero, R_k, n));
        // scale X_k by Lam_k
        CHECK_CUBLAS(hipblasDdgmm(
            cublasH,
            HIPBLAS_SIDE_RIGHT, // scale columns
            n,                 // number of rows
            m,                 // number of columns
            X_k, n,            // input matrix
            Lam_k, 1,         // vector (stride 1)
            X_k_tmp, n       // output matrix
        ));
        // substract it from R_k
        CHECK_CUBLAS(hipblasDaxpy(cublasH, n * m, &neg1, X_k_tmp, 1, R_k, 1));

        CHECK_CUBLAS(hipblasDnrm2(cublasH, n * m, R_k, 1, &norm_R_k));

        if (verbose) {
            std::cout << "LOPBCG iter: " << iter << "||R_k||_F = " << norm_R_k << std::endl;
        }

        // if the norm of R_k is less than tol, break
        if (norm_R_k < tol) {
            if (verbose) {
                std::cout << "Converged: ||R_k||_F < tol" << std::endl;
            }
            break;
        }

        // concatenate X_k, R_k, and Delta_X_k into XRD
        CHECK_CUDA(hipMemcpy(XRD            ,       X_k, n * m * sizeof(double), D2D));
        CHECK_CUDA(hipMemcpy(XRD +     n * m,       R_k, n * m * sizeof(double), D2D));
        CHECK_CUDA(hipMemcpy(XRD + 2 * n * m, Delta_X_k, n * m * sizeof(double), D2D));

        // compute QR factorization of XRD
        CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, n, 3*m, XRD, n, tau_xrd, d_work_xrd, lwork_xrd, devInfo));
        CHECK_CUSOLVER(hipsolverDnDorgqr(cusolverH, n, 3*m, 3*m, XRD, n, tau_xrd, d_work_xrd, lwork_xrd, devInfo));

        // T = Q^T * A * Q
        // T_tmp = Q^T * A
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 3*m, n, n,
                                &one, XRD, n, A, n,
                                &zero, T_tmp_XRD, 3*m));
        // T = T_tmp * Q
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 3*m, 3*m, n,
                                &one, T_tmp_XRD, 3*m, XRD, n,
                                &zero, T_XRD, 3*m));
        // copy T to Tt
        CHECK_CUBLAS(hipblasDcopy(cublasH, 3*m * 3*m, T_XRD, 1, Tt_XRD, 1));

        // T = 0.5 * (T + T^T)
        CHECK_CUBLAS(hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 3*m, 3*m,
                                &half, T_XRD, 3*m,
                                &half, Tt_XRD, 3*m,
                                T_XRD, 3*m));

        // compute eigenvalues and eigenvectors of T
        // both are in increasing order
        CHECK_CUSOLVER(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                        3*m, T_XRD, 3*m, Lam_all, d_work_eig_XRD, lwork_eig_XRD, devInfo));
        // reverse columns of T_XRD
        reverse_columns(T_XRD, T_tmp_XRD, 3*m, 3*m);

        // XRD_tmp = Q * T
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 3*m, 3*m,
                                &one, XRD, n, T_tmp_XRD, 3*m,
                                &zero, XRD_tmp, n));

        // Delta_X_k = - X_k
        CHECK_CUBLAS(hipblasDcopy(cublasH, n * m, X_k, 1, Delta_X_k, 1));
        CHECK_CUBLAS(hipblasDscal(cublasH, n * m, &neg1, Delta_X_k, 1));

        // X_k = XRD_tmp(1:m)
        for (int i = 0; i < m; ++i) {
            CHECK_CUBLAS(hipblasDcopy(cublasH, n, XRD_tmp + i*n, 1, X_k + i*n, 1));
        }

        // Delta = X_kp1 - X_k
        CHECK_CUBLAS(hipblasDaxpy(cublasH, n * m, &one, X_k, 1, Delta_X_k, 1));
        
        // Lam_k = Lam_all(2m:3m)
        CHECK_CUBLAS(hipblasDcopy(cublasH, m, Lam_all + 2*m, 1, Lam_k_tmp, 1));
        reverse_vector(Lam_k_tmp, Lam_k, m);
    }

    /* Copy results to output */
    // V = X_k
    CHECK_CUBLAS(hipblasDcopy(cublasH, n * m, X_k, 1, V, 1));
    // D = Lam_k
    CHECK_CUBLAS(hipblasDcopy(cublasH, m, Lam_k, 1, D, 1));


    // Free device memory
    CHECK_CUDA(hipFree(X_k));
    CHECK_CUDA(hipFree(X_k_tmp));
    CHECK_CUDA(hipFree(Lam_k));
    CHECK_CUDA(hipFree(Lam_k_tmp));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(tau));
    CHECK_CUDA(hipFree(d_work_xrd));
    CHECK_CUDA(hipFree(tau_xrd));
    CHECK_CUDA(hipFree(T_tmp));
    CHECK_CUDA(hipFree(T_tmp_XRD));
    CHECK_CUDA(hipFree(d_work_eig));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipFree(T));
    CHECK_CUDA(hipFree(Tt));
    CHECK_CUDA(hipFree(Delta_X_k));
    CHECK_CUDA(hipFree(R_k));
    
    CHECK_CUDA(hipFree(XRD));
    CHECK_CUDA(hipFree(Lam_all));
    CHECK_CUDA(hipFree(T_XRD));
    CHECK_CUDA(hipFree(Tt_XRD));
    CHECK_CUDA(hipFree(XRD_tmp));
    CHECK_CUDA(hipFree(d_work_eig_XRD));

    CHECK_CUBLAS(hipblasDestroy(cublasH));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
}