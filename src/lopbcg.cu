#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <vector>
#include <cassert>
#include <cstdio>

#include "psd_projection/lopbcg.h"
#include "psd_projection/check.h"
#include "psd_projection/utils.h"

void lopbcg(
    const double* A, // n x n, device pointer
    double* V,       // n x m, device pointer (output eigenvectors)
    double* D,       // m x m, device pointer (output eigenvalues, diagonal)
    const int n,
    const int m ,      // number of eigenpairs
    const int maxiter, // maximum iterations
    const double tol   // convergence tolerance
) {
    assert(m > 0);
    assert(n > 0);
    assert(m <= n);

    /* Allocations */
    // cuBLAS/cuSOLVER handles
    hipblasHandle_t cublasH = nullptr;
    hipsolverHandle_t cusolverH = nullptr;
    CHECK_CUBLAS(hipblasCreate(&cublasH));
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    // allocate the device memory
    double *X_k, *Lam_k, *Q, *T, *Tt, *Delta_X_k, *tmp;
    CHECK_CUDA(hipMalloc(&X_k,       n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_k,         m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Q,         n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T,         m * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Tt,        m * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Delta_X_k, n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&tmp,       n * m * sizeof(double)));

    // workspace for QR decomposition
    int lwork, *devInfo;
    double *d_work, *tau;
    CHECK_CUDA(hipMalloc(&tau, m * sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, n, m, X_k, n, &lwork));
    CHECK_CUDA(hipMalloc(&d_work, lwork * sizeof(double)));
    CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));

    // workspace for eigenvalue decomposition
    int lwork_eig;
    double *d_work_eig;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                                m, T, m, Lam_k, &lwork_eig));
    CHECK_CUDA(hipMalloc(&d_work_eig, lwork_eig * sizeof(double)));

    // useful constants
    const double one = 1.0;
    const double zero = 0.0;
    const double half = 0.5;

    /* Initialization of X_k */
    // initialize X_k with random values on host
    std::vector<double> h_Xk(n * m);
    for (int i = 0; i < n * m; ++i) {
        h_Xk[i] = 2.0 * (static_cast<double>(rand()) / RAND_MAX) - 1.0; // uniform on [-1, 1)
    }
    CHECK_CUDA(hipMemcpy(X_k, h_Xk.data(), n * m * sizeof(double), H2D));

    // compute QR factorization (X_k overwritten with R, tau contains Householder scalars)
    CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, n, m, X_k, n, tau, d_work, lwork, devInfo));

    // generate Q from the result (X_k overwritten with Q)
    CHECK_CUSOLVER(hipsolverDnDorgqr(cusolverH, n, m, m, X_k, n, tau, d_work, lwork, devInfo));

    /* Compute new X_k using T */
    // T = Q^T * A * Q
    // tmp = Q^T * A
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, n,
                             &one, X_k, n, A, n,
                             &zero, tmp, m));
    // T = tmp * Q
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, n,
                             &one, tmp, m, X_k, n,
                             &zero, T, m));
    // copy T to Tt
    CHECK_CUBLAS(hipblasDcopy(cublasH, m * m, T, 1, Tt, 1));

    // T = 0.5 * (T + T^T)
    CHECK_CUBLAS(hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m,
                             &half, T, m,
                             &half, Tt, m,
                             T, m));

    // compute eigenvalues and eigenvectors of T
    // both are in increasing order
    CHECK_CUSOLVER(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                    m, T, m, Lam_k, d_work_eig, lwork_eig, devInfo));

    // X_k = Q * T
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                             &one, X_k, n, T, m,
                             &zero, X_k, n));

    

    // Free device memory
    CHECK_CUDA(hipFree(X_k));
    CHECK_CUDA(hipFree(Lam_k));
    CHECK_CUDA(hipFree(Q));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(tau));
    CHECK_CUDA(hipFree(d_work_eig));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipFree(T));
    CHECK_CUDA(hipFree(Tt));
    CHECK_CUDA(hipFree(Delta_X_k));
    CHECK_CUBLAS(hipblasDestroy(cublasH));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
}