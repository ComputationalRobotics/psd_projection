#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <vector>
#include <cassert>
#include <cstdio>

#include "psd_projection/lopbcg.h"
#include "psd_projection/check.h"
#include "psd_projection/utils.h"

void lopbcg(
    const double* A, // n x n, device pointer
    double* V,       // n x m, device pointer (output eigenvectors)
    double* D,       // m x m, device pointer (output eigenvalues, diagonal)
    const int n,
    const int m ,      // number of eigenpairs
    const int maxiter, // maximum iterations
    const double tol   // convergence tolerance
) {
    assert(m > 0);
    assert(n > 0);
    assert(3*m <= n);

    /* Allocations */
    // cuBLAS/cuSOLVER handles
    hipblasHandle_t cublasH = nullptr;
    hipsolverHandle_t cusolverH = nullptr;
    CHECK_CUBLAS(hipblasCreate(&cublasH));
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    // allocate the device memory
    double *X_k, *Lam_k, *T, *Tt, *Delta_X_k, *T_tmp, *R_k;
    double *XRD, *Lam_all, *XRD_tmp, *T_XRD, *Tt_XRD, *T_tmp_XRD;
    CHECK_CUDA(hipMalloc(&X_k,            n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_k,              m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Delta_X_k,      n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_tmp,          n * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T,              m * m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Tt,             m * m * sizeof(double)));
    
    CHECK_CUDA(hipMalloc(&XRD,          n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Lam_all,          3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&XRD_tmp,      n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_XRD,      3*m * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&Tt_XRD,     3*m * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&T_tmp_XRD,    n * 3*m * sizeof(double)));
    CHECK_CUDA(hipMalloc(&R_k,          n * 3*m * sizeof(double)));

    double norm_R_k;

    // workspace for QR decomposition of X_k
    int lwork, *devInfo;
    double *d_work, *tau;
    CHECK_CUDA(hipMalloc(&tau, m * sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, n, m, X_k, n, &lwork));
    CHECK_CUDA(hipMalloc(&d_work, lwork * sizeof(double)));
    CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));

    // workspace for QR decomposition of XRD
    int lwork_xrd;
    double *d_work_xrd, *tau_xrd;
    CHECK_CUDA(hipMalloc(&tau_xrd, 3*m * sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDgeqrf_bufferSize(cusolverH, n, 3*m, XRD, n, &lwork_xrd));
    CHECK_CUDA(hipMalloc(&d_work_xrd, lwork_xrd * sizeof(double)));

    // workspace for eigenvalue decomposition
    int lwork_eig;
    double *d_work_eig;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                                m, T, m, Lam_k, &lwork_eig));
    CHECK_CUDA(hipMalloc(&d_work_eig, lwork_eig * sizeof(double)));

    // workspace for eigenvalue decomposition of XRD
    int lwork_eig_XRD;
    double *d_work_eig_XRD;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                                3*m, T_XRD, 3*m, Lam_all, &lwork_eig_XRD));
    CHECK_CUDA(hipMalloc(&d_work_eig_XRD, lwork_eig_XRD * sizeof(double)));

    // useful constants
    const double one = 1.0;
    const double zero = 0.0;
    const double half = 0.5;
    const double neg1 = -1.0;

    /* Initialization of X_k */
    // initialize X_k with random values on host
    std::vector<double> h_Xk(n * m);
    for (int i = 0; i < n * m; ++i) {
        h_Xk[i] = 2.0 * (static_cast<double>(rand()) / RAND_MAX) - 1.0; // uniform on [-1, 1)
    }
    CHECK_CUDA(hipMemcpy(X_k, h_Xk.data(), n * m * sizeof(double), H2D));

    // compute QR factorization (X_k overwritten with R, tau contains Householder scalars)
    CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, n, m, X_k, n, tau, d_work, lwork, devInfo));

    // generate Q from the result (X_k overwritten with Q)
    CHECK_CUSOLVER(hipsolverDnDorgqr(cusolverH, n, m, m, X_k, n, tau, d_work, lwork, devInfo));

    /* Compute new X_k using T */
    // T = Q^T * A * Q
    // T_tmp = Q^T * A
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, n,
                             &one, X_k, n, A, n,
                             &zero, T_tmp, m));
    // T = T_tmp * Q
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, n,
                             &one, T_tmp, m, X_k, n,
                             &zero, T, m));
    // copy T to Tt
    CHECK_CUBLAS(hipblasDcopy(cublasH, m * m, T, 1, Tt, 1));

    // T = 0.5 * (T + T^T)
    CHECK_CUBLAS(hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m,
                             &half, T, m,
                             &half, Tt, m,
                             T, m));

    // compute eigenvalues and eigenvectors of T
    // both are in increasing order
    CHECK_CUSOLVER(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                    m, T, m, Lam_k, d_work_eig, lwork_eig, devInfo));

    // X_k = Q * T
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                             &one, X_k, n, T, m,
                             &zero, X_k, n));

    // Delta_X_k = X_k
    CHECK_CUBLAS(hipblasDcopy(cublasH, n * m, X_k, 1, Delta_X_k, 1));

    for (int iter = 1; iter <= maxiter; iter++) {
        // R_k = A * X_k - X_k * Lam_k
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                                 &one, A, n, X_k, n,
                                 &zero, R_k, n));
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                                 &neg1, X_k, n, Lam_k, m,
                                 &one, R_k, n));

        // if the norm of R_k is less than tol, break
        CHECK_CUBLAS(hipblasDnrm2(cublasH, n * m, R_k, 1, &norm_R_k));
        if (norm_R_k < tol)
            break;

        // concatenate X_k, R_k, and Delta_X_k into XRD
        CHECK_CUDA(hipMemcpy(XRD            ,       X_k, n * m * sizeof(double), D2D));
        CHECK_CUDA(hipMemcpy(XRD +     n * m,       R_k, n * m * sizeof(double), D2D));
        CHECK_CUDA(hipMemcpy(XRD + 2 * n * m, Delta_X_k, n * m * sizeof(double), D2D));

        // compute QR factorization of XRD
        CHECK_CUSOLVER(hipsolverDnDgeqrf(cusolverH, n, 3*m, XRD, n, tau_xrd, d_work_xrd, lwork_xrd, devInfo));
        CHECK_CUSOLVER(hipsolverDnDorgqr(cusolverH, n, 3*m, 3*m, XRD, n, tau_xrd, d_work_xrd, lwork_xrd, devInfo));

        // T = Q^T * A * Q
        // T_tmp = Q^T * A
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 3*m, n, n,
                                &one, XRD, n, A, n,
                                &zero, T_tmp_XRD, 3*m));
        // T = T_tmp * Q
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 3*m, 3*m, n,
                                &one, T_tmp_XRD, 3*m, X_k, n,
                                &zero, T_XRD, 3*m));
        // copy T to Tt
        CHECK_CUBLAS(hipblasDcopy(cublasH, 3*m * 3*m, T_XRD, 1, Tt_XRD, 1));

        // T = 0.5 * (T + T^T)
        CHECK_CUBLAS(hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 3*m, 3*m,
                                &half, T_XRD, 3*m,
                                &half, Tt_XRD, 3*m,
                                T_XRD, 3*m));

        // compute eigenvalues and eigenvectors of T
        // both are in increasing order
        CHECK_CUSOLVER(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                        3*m, T_XRD, 3*m, Lam_all, d_work_eig_XRD, lwork_eig_XRD, devInfo));

        // XRD_tmp = Q * T
        CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, m,
                                &one, XRD_tmp, n, T, m,
                                &zero, XRD_tmp, n));
    }


    // Free device memory
    CHECK_CUDA(hipFree(X_k));
    CHECK_CUDA(hipFree(Lam_k));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(tau));
    CHECK_CUDA(hipFree(d_work_xrd));
    CHECK_CUDA(hipFree(tau_xrd));
    CHECK_CUDA(hipFree(T_tmp));
    CHECK_CUDA(hipFree(T_tmp_XRD));
    CHECK_CUDA(hipFree(d_work_eig));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipFree(T));
    CHECK_CUDA(hipFree(Tt));
    CHECK_CUDA(hipFree(Delta_X_k));
    CHECK_CUDA(hipFree(R_k));
    
    CHECK_CUDA(hipFree(XRD));
    CHECK_CUDA(hipFree(Lam_all));
    CHECK_CUDA(hipFree(T_XRD));
    CHECK_CUDA(hipFree(Tt_XRD));
    CHECK_CUDA(hipFree(XRD_tmp));
    CHECK_CUDA(hipFree(d_work_eig_XRD));

    CHECK_CUBLAS(hipblasDestroy(cublasH));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
}