#include "hip/hip_runtime.h"
/*

    psd_projection_MATLAB.cu

    This file is part of psd_projection. It defines MATLAB interface functions for the psd_projection library.

*/

#include <memory>
#include <vector>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <cassert>

#include "mex.h"
#include "matrix.h"
#include "mat.h"

#include "psd_projection/check.h"
#include "psd_projection/utils.h"
#include "psd_projection/composite_FP32.h"
#include "psd_projection/composite_TF16.h"
#include "psd_projection/haoyu_TF16.h"
#include "psd_projection/lanczos.h"

void get_dnmat_from_matlab(
    const mxArray* mx_dnmat,
    size_t* n,
    std::vector<double>& cpu_dnmat_vals
) {
    // read the matrix size from MATLAB and check that it is square
    int cpu_dnmat_row_size = static_cast<int>( mxGetM(mx_dnmat) );
    int cpu_dnmat_col_size = static_cast<int>( mxGetN(mx_dnmat) );
    assert(cpu_dnmat_row_size == cpu_dnmat_col_size);
    *n = static_cast<size_t>(cpu_dnmat_row_size);

    double* cpu_dnmat_vals_pointer = mxGetPr(mx_dnmat);
    cpu_dnmat_vals.clear();
    cpu_dnmat_vals.resize(*n * *n, 0);
    memcpy(cpu_dnmat_vals.data(), cpu_dnmat_vals_pointer, sizeof(double) * *n * *n);
    return;
}

// input order
class INPUT_ID_factory {
    public:
        int mat;
        int method;

        INPUT_ID_factory(int offset = 0) {
            this->mat = offset + 0;
            this->method = offset + 1;
        }
};

void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    /* Input */
    INPUT_ID_factory INPUT_ID(0);
    if (nrhs != 2) {
        mexErrMsgTxt("Wrong number of input arguments. Expected 2 inputs: mat, method.");
    }
    
    // get the matrix
    size_t n;
    std::vector<double> cpu_At_csc_vals;
    get_dnmat_from_matlab(prhs[INPUT_ID.mat], &n, cpu_At_csc_vals);

    // get the method
    const mxArray* mx_method = prhs[INPUT_ID.method];
    if (!mxIsChar(mx_method)) {
        mexErrMsgTxt("The 'method' input must be a string.");
    }
    char* method_cstr = mxArrayToString(mx_method);
    if (!method_cstr) {
        mexErrMsgTxt("Failed to convert 'method' input to string.");
    }
    std::string method(method_cstr);
    mxFree(method_cstr);

    /* Project the matrix */
    // create the handles
    hipsolverHandle_t solverH;
    CHECK_CUSOLVER(hipsolverDnCreate(&solverH));

    hipblasHandle_t cublasH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));
    if (method == "haoyu_TF16" || method == "composite_TF16") {
        CHECK_CUBLAS(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));
    }
    
    // create the host matrix
    double *dA_psd;
    CHECK_CUDA(hipMalloc(&dA_psd, n * n * sizeof(double)));
    CHECK_CUDA(hipMemcpy(dA_psd, cpu_At_csc_vals.data(), n * n * sizeof(double), H2D));

    // approximate the spectral norm
    double lo, up;
    approximate_two_norm(
        cublasH, solverH, dA_psd, n, &lo, &up
    );

    // scale to have eigenvalues in [-1, 1]
    const double scale = up > 0.0 ? up : 1.0;
    const double inv_scale = 1.0/scale;
    CHECK_CUBLAS( hipblasDscal(cublasH, n*n, &inv_scale, dA_psd, 1) );

    // call the appropriate method
    if (method == "composite_TF16")
        composite_TF16(cublasH, dA_psd, n);
    else if (method == "composite_FP32")
        composite_FP32(cublasH, dA_psd, n);
    else if (method == "haoyu_TF16") {
        float* dA_psd_float;
        CHECK_CUDA(hipMalloc(&dA_psd_float, n * n * sizeof(float)));
        convert_double_to_float(dA_psd, dA_psd_float, n * n);
        haoyu_TF16(cublasH, dA_psd_float, n);
        convert_float_to_double(dA_psd_float, dA_psd, n * n);
        CHECK_CUDA(hipFree(dA_psd_float));
     }
    else {
        mexErrMsgTxt("Unknown method. Supported methods: 'composite_TF16', 'composite_FP32', 'haoyu_TF16'.");
        return;
    }

    // unscale
    CHECK_CUBLAS( hipblasDscal(cublasH, n*n, &scale, dA_psd, 1) );
    CHECK_CUDA(hipDeviceSynchronize());

    /* Output the result */
    plhs[0] = mxCreateDoubleMatrix(n, n, mxREAL);
    double* cpu_At_psd_vals = mxGetPr(plhs[0]);
    CHECK_CUDA(hipMemcpy(cpu_At_psd_vals, dA_psd, n * n * sizeof(double), D2H));

    // free
    CHECK_CUDA(hipFree(dA_psd));
    CHECK_CUBLAS(hipblasDestroy(cublasH));
    CHECK_CUSOLVER(hipsolverDnDestroy(solverH));
}