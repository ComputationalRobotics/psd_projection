#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size_M>" << std::endl;
        return -1;
    }

    int M = std::atoi(argv[1]);
    int N = M;
    int K = M;
    size_t size_A = static_cast<size_t>(M) * K;
    size_t size_B = static_cast<size_t>(K) * N;
    size_t size_C = static_cast<size_t>(M) * N;

    // Host allocations
    std::vector<float> h_A(size_A);
    std::vector<float> h_B(size_B);
    std::vector<float> h_C(size_C, 0.0f);

    // Random fill
    std::mt19937 gen(42);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (auto &v : h_A) v = dist(gen);
    for (auto &v : h_B) v = dist(gen);

    // Device allocations
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A * sizeof(float));
    hipMalloc(&d_B, size_B * sizeof(float));
    hipMalloc(&d_C, size_C * sizeof(float));

    hipMemcpy(d_A, h_A.data(), size_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, size_C * sizeof(float));

    // Create cuBLAS handle and enable Tensor Core math
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    cublasSetEmulationStrategy(handle, CUBLAS_EMULATION_STRATEGY_EAGER);

    float alpha = 1.0f;
    float beta  = 0.0f;

    // Warm-up to exclude startup overhead
    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        d_B, HIP_R_32F, N,
        d_A, HIP_R_32F, K,
        &beta,
        d_C, HIP_R_32F, N,
        CUBLAS_COMPUTE_32F_EMULATED_16BFX9,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );
    hipDeviceSynchronize();

    // Setup CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start
    hipEventRecord(start);

    // Actual GEMM
    for(int i = 0; i < 100; ++i) {
        hipblasGemmEx(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, M, K,
            &alpha,
            d_B, HIP_R_32F, N,
            d_A, HIP_R_32F, K,
            &beta,
            d_C, HIP_R_32F, N,
            CUBLAS_COMPUTE_32F_EMULATED_16BFX9,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP
        );
    }

    // Record stop and synchronize
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    std::cout << "Tensor-Core GEMM took " << elapsed_ms << " ms (excluding warm-up)" << std::endl;

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Tensor-Core GEMM failed" << std::endl;
        return -1;
    }

    // Copy result back
    hipMemcpy(h_C.data(), d_C, size_C * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first element as a sanity check
    std::cout << "C[0,0] = " << h_C[0] << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}